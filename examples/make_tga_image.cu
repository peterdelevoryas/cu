#include "hip/hip_runtime.h"
fn printf(fmt: *i8, ...);
fn malloc(size: i64) -> *i8;
fn memset(b: *i8, c: i32, len: i64) -> *i8;
fn memcpy(dst: *i8, src: *i8, n: i64) -> *i8;
fn free(ptr: *i8);
type FILE = ();
fn fopen(path: *i8, mode: *i8) -> *FILE;
fn fwrite(ptr: *i8, size: i64, nitems: i64, stream: *FILE) -> i64;
fn fclose(stream: *FILE) -> i32;

type point struct {
    x: i32,
    y: i32,
}

type pixel struct {
    b: i8,
    g: i8,
    r: i8,
}

type image struct {
    pixels: *pixel,
    width: i32,
    height: i32,
}

fn new_image(width: i32, height: i32) -> image {
    let size = width as i64 * height as i64 * sizeof(pixel);
    let pixels = malloc(size);
    memset(pixels, 0, size);
    let pixels = pixels as *pixel;
    return {
        pixels: pixels,
        width: width,
        height: height,
    };
}

fn set_pixel(image: image, p: point, color: pixel) {
    if p.x >= image.width {
        return;
    }
    if p.y >= image.height {
        return;
    }
    image.pixels[image.width * p.y + p.x] = color;
}

fn abs(x: i32) -> i32 {
    if x < 0 {
        return -1 * x;
    }
    return x;
}

fn swap_i32(x: *i32, y: *i32) {
    let tmp = *x;
    *x = *y;
    *y = tmp;
}

fn swap_points(x: *point, y: *point) {
    let tmp = *x;
    *x = *y;
    *y = tmp;
}

fn draw_line(image: image, a: point, b: point, color: pixel) {
    let steep = false;
    if abs(a.x - b.x) < abs(a.y - b.y) {
        swap_i32(&a.x, &a.y);
        swap_i32(&b.x, &b.y);
        steep = true;
    }
    if a.x > b.x {
        swap_points(&a, &b);
    }
    let m = (b.y - a.y) as f32 / (b.x - a.x) as f32;
    for let x = a.x; x <= b.x; x += 1 {
        let dx = (x - a.x) as f32;
        let y = m * dx + a.y as f32;
        let p: point = { x: x, y: y as i32 };
        if steep {
            swap_i32(&p.x, &p.y);
        }
        set_pixel(image, p, color);
    }
}

fn draw_grid(image: image) {
    let spacing = 50;
    for let x = spacing; x < image.width; x += spacing {
        for let y = spacing; y < image.height; y += spacing {
            draw_line(image, { x: x, y: 0 }, { x: x, y: image.height }, { b: 0, g: 0, r: 255 });
            draw_line(image, { x: 0, y: y }, { x: image.width, y: y }, { b: 0, g: 0, r: 255 });
        }
    }
}

fn flip_image(image: image) {
    let row_size = image.width as i64 * sizeof(pixel);
    let tmp = malloc(row_size);

    for let row = 0; row < image.height / 2; row += 1 {
        let top_row = &image.pixels[row * image.width] as *i8;
        let bottom_row = &image.pixels[(image.height - 1 - row) * image.width] as *i8;
        memcpy(tmp, top_row, row_size);
        memcpy(top_row, bottom_row, row_size);
        memcpy(bottom_row, tmp, row_size);
    }

    free(tmp);
}

fn split_i16(x: i16) -> (i8, i8)  {
    let lower = ((x >> 0) & 255) as i8;
    let upper = ((x >> 8) & 255) as i8;
    return (lower, upper);
}

fn write_tga_file(image: image, path: *i8) {
    let pixels_size = image.width as i64 * image.height as i64 * sizeof(pixel);
    let header_size: i64 = 18;
    let output_size = header_size + pixels_size;
    let output = malloc(output_size);

    memset(output, 0, header_size);
    output[2] = 2;
    let (x, y) = split_i16(image.width as i16);
    output[12] = x;
    output[13] = y;
    let (x, y) = split_i16(image.height as i16);
    output[14] = x;
    output[15] = y;
    output[16] = (sizeof(pixel) * 8) as i8;
    output[17] = 32;

    memcpy(output + header_size as i32, image.pixels as *i8, pixels_size);

    let f = fopen(path, "w");
    fwrite(output, 1, output_size, f);
    fclose(f);
    free(output);
}

fn main() {
    let image = new_image(200, 200);
    draw_grid(image);
    flip_image(image);
    write_tga_file(image, "output.tga");
}
